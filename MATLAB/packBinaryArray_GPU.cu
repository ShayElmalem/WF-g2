#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <stdint.h> // Include this header for uint8_t

#define BITS_PER_BYTE 8

// CUDA kernel to pack binary values into uint8
__global__ void packBinaryKernel(const uint8_t *input, uint8_t *output, size_t M, size_t N, size_t K) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < M && y < N && z < (K / BITS_PER_BYTE)) {
        uint8_t packedValue = 0;

        for (int bit = 0; bit < BITS_PER_BYTE; ++bit) {
            int originalZ = z * BITS_PER_BYTE + bit;
            uint8_t bitValue = input[x + y * M + originalZ * M * N];
            packedValue |= (bitValue << bit);
        }

        output[x + y * M + z * M * N] = packedValue;
    }
}

// MEX gateway function
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    // Initialize the GPU API
    mxInitGPU();

    // Check for proper number of arguments
    if (nrhs != 1) {
        mexErrMsgIdAndTxt("packBinaryArray_GPU:invalidNumInputs", "One input required.");
    }
    if (nlhs != 1) {
        mexErrMsgIdAndTxt("packBinaryArray_GPU:invalidNumOutputs", "One output required.");
    }

    // Get input 3D array
    const mxGPUArray *input = mxGPUCreateFromMxArray(prhs[0]);

    // Ensure the input is a uint8 array
    if (mxGPUGetClassID(input) != mxUINT8_CLASS) {
        mexErrMsgIdAndTxt("packBinaryArray_GPU:invalidInput", "Input must be a uint8 array.");
    }

    const mwSize *dims = mxGPUGetDimensions(input);
    size_t M = dims[0];
    size_t N = dims[1];
    size_t K = dims[2];

    if (K % BITS_PER_BYTE != 0) {
        mexErrMsgIdAndTxt("packBinaryArray_GPU:invalidDimension", "The third dimension (K) must be divisible by 8.");
    }

    // Create output 3D array
    mwSize outputDims[3] = {M, N, K / BITS_PER_BYTE};
    mxGPUArray *output = mxGPUCreateGPUArray(3, outputDims, mxUINT8_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    const uint8_t *inputData = (const uint8_t *)mxGPUGetDataReadOnly(input);
    uint8_t *outputData = (uint8_t *)mxGPUGetData(output);

    // Define grid and block sizes
    dim3 blockDim(8, 8, 8);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y, (K / BITS_PER_BYTE + blockDim.z - 1) / blockDim.z);

    // Launch the CUDA kernel
    packBinaryKernel<<<gridDim, blockDim>>>(inputData, outputData, M, N, K);

    // Synchronize the device
    hipDeviceSynchronize();

    // Return the output
    plhs[0] = mxGPUCreateMxArrayOnGPU(output);

    // Destroy GPU arrays
    mxGPUDestroyGPUArray(input);
    mxGPUDestroyGPUArray(output);
}
