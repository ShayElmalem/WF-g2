#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>

// Define the lookup table for counting the number of ones in a byte
__device__ __constant__ unsigned char lookupTable[256] = {
    0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 4,
    1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
    1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
    1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
    2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
    3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
    3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
    4, 5, 5, 6, 5, 6, 6, 7, 5, 6, 6, 7, 6, 7, 7, 8
};

// Kernel function to process each element using the lookup table
__global__ void countOnesKernel(const unsigned char* input, unsigned char* output, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = lookupTable[input[idx]];
    }
}

// Entry point for the MEX function
void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    // Initialize the GPU API
    mxInitGPU();

    // Get the input gpuArray
    mxGPUArray const *inputGPUArray = mxGPUCreateFromMxArray(prhs[0]);
    const unsigned char *input = (const unsigned char *)(mxGPUGetDataReadOnly(inputGPUArray));

    // Get the dimensions of the input array
    mwSize const *dims = mxGPUGetDimensions(inputGPUArray);
    size_t N = mxGPUGetNumberOfElements(inputGPUArray);

    // Create the output gpuArray
    mxGPUArray *outputGPUArray = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(inputGPUArray), 
                                                     dims, 
                                                     mxUINT8_CLASS, 
                                                     mxREAL, 
                                                     MX_GPU_DO_NOT_INITIALIZE);
    unsigned char *output = (unsigned char *)(mxGPUGetData(outputGPUArray));

    // Set up the execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    countOnesKernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);

    // Wrap the output gpuArray into an mxArray and return it
    plhs[0] = mxGPUCreateMxArrayOnGPU(outputGPUArray);

    // Destroy the input and output gpuArray
    mxGPUDestroyGPUArray(inputGPUArray);
    mxGPUDestroyGPUArray(outputGPUArray);
}
